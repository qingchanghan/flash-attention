#include "hip/hip_runtime.h"
#include <assert.h>
#include <c10/macros/Macros.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <stdint.h>

#include <cfloat>
#include <limits>

#include "common.cuh"
#include "cuda_utils.h"
#include "kernels.h"

namespace {

template <typename T>
__global__ void reduce_sum_ker_v0(const T* input, T* output, const int rows,
                                  const int cols) {
  int row_idx = blockIdx.x;
  if (row_idx >= rows) return;
  const T* cur_input = input + row_idx * cols;
  T local_sum = (T)0.;
  for (int i = threadIdx.x; i < cols; i += blockDim.x) {
    local_sum = local_sum + cur_input[i];
  }

  local_sum = block_reduce_sum<T>(local_sum);
  if (threadIdx.x == 0) {
    output[row_idx] = local_sum;
  }
}
}  // namespace

template <typename T>
void reduce_sum_cuda_v0(const T* input, T* output, const int rows,
                        const int cols, hipStream_t stream) {
  reduce_sum_ker_v0<T><<<rows, 1024, 0, stream>>>(input, output, rows, cols);
}

#define INSTANTIATE_REDUCE_SUM_V0(T)                                          \
  template void reduce_sum_cuda_v0(const T* input, T* output, const int rows, \
                                   const int cols, hipStream_t stream);

INSTANTIATE_REDUCE_SUM_V0(float)
INSTANTIATE_REDUCE_SUM_V0(half)
INSTANTIATE_REDUCE_SUM_V0(__hip_bfloat16)
